
#include <hip/hip_runtime.h>

__global__ void blur(unsigned char* input_image, unsigned char* output_image, int const width, int const height) {

	unsigned int const pixel_offset = blockIdx.x * blockDim.x + threadIdx.x;

	int const x = pixel_offset % width;
	int const y = pixel_offset / width;
	int const filter_size = 5;
	if (pixel_offset < (width * height)) {
		float output_red = 0;
		float output_green = 0;
		float output_blue = 0;
		int hits = 0;
		for (int ox = -filter_size; ox <= filter_size; ++ox) {
			for (int oy = -filter_size; oy <= filter_size; ++oy) {
				if ((x + ox) >= 0 && (x + ox) < width && (y + oy) >= 0 && (y + oy) < height) {
					int const color_offset = (pixel_offset + ox + oy * width) * 3;
					output_red += input_image[color_offset];
					output_green += input_image[color_offset + 1];
					output_blue += input_image[color_offset + 2];
					++hits;
				}
			}
		}
		output_image[pixel_offset * 3] = output_red / hits;
		output_image[pixel_offset * 3 + 1] = output_green / hits;
		output_image[pixel_offset * 3 + 2] = output_blue / hits;
	}
}

void filter(unsigned char* input_image, unsigned char* output_image, unsigned int const width, unsigned int const height) {
	unsigned char *dev_input;
	unsigned char *dev_output;

	unsigned int const size = width * height * 3;

	hipMallocManaged(reinterpret_cast<void **>(&dev_input), size * sizeof(unsigned char));
	hipMallocManaged(reinterpret_cast<void **>(&dev_output), size * sizeof(unsigned char));

	memcpy(dev_input, input_image, size * sizeof(unsigned char));

	hipDeviceSynchronize();

	dim3 blockDims { 128 };
	dim3 gridDims { (width * height + 127) / 128 };

	blur<<< gridDims, blockDims >>>(dev_input, dev_output, width, height);

	hipDeviceSynchronize();

	memcpy(output_image, dev_output, size * sizeof(unsigned char));

	hipFree(dev_input);
	hipFree(dev_output);
}
